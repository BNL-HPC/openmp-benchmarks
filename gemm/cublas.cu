#define CATCH_CONFIG_ENABLE_BENCHMARKING
#ifndef CATCH_CONFIG_ENABLE_BENCHMARKING 
#endif

#include <cstdlib>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cuda_bench.cuh>
#include <catch2/catch_test_macros.hpp>
#include <catch2/benchmark/catch_benchmark.hpp>
#include <common.hpp>

namespace cuda_bench {

template double* cublas_wrapper_d <double> ( const int, const int, const int);
template float*  cublas_wrapper_f <float>  ( const int, const int, const int);
//template int*    sgemm_wrapper <int>    ( const int, const int, const int);


// matrix multiplication C = alpha * A * B + beta * C
template<typename T>
__host__ T* cublas_wrapper_f ( const int M, const int N, const int K ) {

  hipError_t cudaStat;  // hipMalloc status
  hipblasStatus_t stat;   // cuBLAS functions status

  T* h_A = (T*)malloc(sizeof(T) * M * K);
  T* h_B = (T*)malloc(sizeof(T) * K * N);
  T* h_C = (T*)malloc(sizeof(T) * M * N);
  
  srand(time(0));
  T epsilon = common::get_epsilon <T> ();
  for (int i = 0; i < M*K; i++)
    h_A[i] = common::initialize_random ( epsilon );

  for (int i = 0; i < K*N; i++)
    h_B[i] = common::initialize_random ( epsilon );

  for (int i = 0; i < M*N; i++)
    h_C[i] = common::initialize_random ( epsilon );
 
  BENCHMARK_ADVANCED("cuBLAS SGEMM")(Catch::Benchmark::Chronometer meter) {

    T *d_A, *d_B, *d_C;
    hipMalloc( (void**) &d_A, sizeof(T) * M * K);
    hipMalloc( (void**) &d_B, sizeof(T) * K * N);
    hipMalloc( (void**) &d_C, sizeof(T) * M * N);
    hipblasHandle_t handle; // cuBLAS context

    stat = hipblasCreate(&handle); // initialize CUBLAS context

    hipMemcpy(d_A, h_A, M * K * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(T), hipMemcpyHostToDevice);

    T alpha(1.0), beta(0.5);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    meter.measure( [handle, N, M, K, &alpha, &beta, d_B, d_A, d_C] { 
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N,
                     d_A, K, &beta, d_C, N); 
    hipDeviceSynchronize() ;
    });
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msecTotal(0.0f);
    hipEventElapsedTime(&msecTotal, start, stop);
    hipMemcpy(h_C, d_C, M * N * sizeof(T), hipMemcpyDeviceToHost);

    double Gflops = 2.0 * M * N * K * 1e-9;
    double avg_time = msecTotal*1000000.0;
    //printf("Average elapsed time: (%7.6f) ns %f, %f, %f \n", avg_time, h_A[0], h_B[0], h_C[0]);
    //printf("Gflops = %f \n", Gflops);
    //printf("%f GFLOPs/s \n", Gflops/avg_time);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle); // destroy CUBLAS context
  };
  free(h_A);
  free(h_B);
  free(h_C);

  return h_A; 
}


template<typename T>
__host__ T* cublas_wrapper_d ( const int M, const int N, const int K ) {

  hipError_t cudaStat;  // hipMalloc status
  hipblasStatus_t stat;   // cuBLAS functions status
  hipblasHandle_t handle; // cuBLAS context

  T* h_A = (T*)malloc(sizeof(T) * M * K);
  T* h_B = (T*)malloc(sizeof(T) * K * N);
  T* h_C = (T*)malloc(sizeof(T) * M * N);
  
  srand(time(0));
  T epsilon = common::get_epsilon <T> ();
  for (int i = 0; i < M*K; i++)
    h_A[i] = common::initialize_random ( epsilon );

  for (int i = 0; i < K*N; i++)
    h_B[i] = common::initialize_random ( epsilon );

  for (int i = 0; i < M*N; i++)
    h_C[i] = common::initialize_random ( epsilon );
 
  BENCHMARK_ADVANCED("cuBLAS SGEMM")(Catch::Benchmark::Chronometer meter) {

    T *d_A, *d_B, *d_C;
    hipMalloc( (void**) &d_A, sizeof(T) * M * K);
    hipMalloc( (void**) &d_B, sizeof(T) * K * N);
    hipMalloc( (void**) &d_C, sizeof(T) * M * N);

    stat = hipblasCreate(&handle); // initialize CUBLAS context

    hipMemcpy(d_A, h_A, M * K * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(T), hipMemcpyHostToDevice);

    T alpha(1.0), beta(0.5);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    meter.measure( [handle, N, M, K, &alpha, &beta, d_B, d_A, d_C] { 
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N,
                     d_A, K, &beta, d_C, N); 
    hipDeviceSynchronize() ;
    });
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msecTotal(0.0f);
    hipEventElapsedTime(&msecTotal, start, stop);
    hipMemcpy(h_C, d_C, M * N * sizeof(T), hipMemcpyDeviceToHost);

    double Gflops = 2.0 * M * N * K * 1e-9;
    double avg_time = msecTotal*1000000.0;
    //printf("Average elapsed time: (%7.6f) ns %f, %f, %f \n", avg_time, h_A[0], h_B[0], h_C[0]);
    //printf("Gflops = %f \n", Gflops);
    //printf("%f GFLOPs/s \n", Gflops/avg_time);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle); // destroy CUBLAS context
  };
  free(h_A);
  free(h_B);
  free(h_C);

  return h_A; 
}



} // namespace cuda_bench
