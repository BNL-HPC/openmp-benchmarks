#include "hip/hip_runtime.h"
#ifndef CATCH_CONFIG_ENABLE_BENCHMARKING 
#define CATCH_CONFIG_ENABLE_BENCHMARKING
#endif

#include <catch.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_bench.cuh>
#include <iostream>
#include <cstdlib>

namespace cuda_bench {

template double* set_to_zero_wrapper <double> ( const int, const int );
template float*  set_to_zero_wrapper <float>  ( const int, const int );
template int*    set_to_zero_wrapper <int>    ( const int, const int );
	
template <typename T>
__global__ void set_to_zero( T* cuda_dev_array, const int N ) {

  unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;
  cuda_dev_array[tid] = 0.0; 
  return;
}

template <>
__global__ void set_to_zero <int> ( int* cuda_dev_array, const int N ) {

  unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;
  cuda_dev_array[tid] = 0; 
  return;
}

template <typename T>
__host__ T* set_to_zero_wrapper ( const int N, const int blocksize ) {

  T *data = (T *) malloc( N * sizeof( T ) );
  for(int i=0; i<N; i++)
  {
    data[i] = drand48();
  }

  /* Allocate an array of length N */
  T* cuda_dev_array; 
  hipMalloc((void**)&cuda_dev_array, sizeof( T ) * N);
  
  hipMemcpy(cuda_dev_array, data, N * sizeof( T ), hipMemcpyHostToDevice);


  int threads_tot = N;
  int nblocks     = ( threads_tot + blocksize - 1 ) / blocksize;

  //set_to_zero<<<nblocks, blocksize>>>(cuda_dev_array, N );
  BENCHMARK("CUDA Array Init") { return set_to_zero<<<nblocks, blocksize>>>( cuda_dev_array, N ); };
  //BENCHMARK_ADVANCED("CUDA Array Init")(Catch::Benchmark::Chronometer meter) {
  //  meter.measure([cuda_dev_array, N, nblocks, blocksize] 
  //  { return set_to_zero<<<nblocks, blocksize>>>( cuda_dev_array, N ); });
  //  hipDeviceSynchronize() ;
  //};
  
  T *cuda_host_array = (T *) malloc( N * sizeof( T ) );
  hipMemcpy(cuda_host_array, cuda_dev_array, N * sizeof( T ), hipMemcpyDeviceToHost);

  bool test_flag = true;
  for ( int i = 0; i < N; i++ ) {
       if ( std::fabs ( cuda_host_array[i] ) > 1e-20 ) 
	  //std::cout << "!!Problem at i = " << i << std::endl;
	  test_flag = false;
  }
  
  //REQUIRE(test_flag == true);
  CHECK(test_flag == true);

  hipFree( cuda_dev_array );
  free( cuda_host_array);

  return cuda_dev_array;
}

} // namespace cuda_bench
